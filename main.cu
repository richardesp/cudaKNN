#include <cstdio>
#include <cmath>
#include "include/Dataset.h"
#include "include/cuda/Lock.h"
#include "include/cuda/cudaKNN.cuh"
#include "include/Point.h"
#include "include/Label.h"


int main(int argc, char **argv) {
    int deviceCount = 0;

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        fprintf(stderr, "There is no device supporting CUDA\n");
        return EXIT_FAILURE;
    }

    int bestDevice = 0;
    hipDeviceProp_t bestDeviceProp;

    for (int i = 0; i < deviceCount; i++) {

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        fprintf(stdout, "\tdevice %d: %s\n", i, deviceProp.name);
        fprintf(stdout, "\tCompute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        fprintf(stdout, "\tGlobal memory: %ld\n", deviceProp.totalGlobalMem);
        fprintf(stdout, "\tShared memory per block: %ld\n", deviceProp.sharedMemPerBlock);
        fprintf(stdout, "\tRegisters per block: %d\n", deviceProp.regsPerBlock);
        fprintf(stdout, "\tWarp size: %d\n", deviceProp.warpSize);
        fprintf(stdout, "\tMemory pitch: %ld\n", deviceProp.memPitch);
        fprintf(stdout, "\tMax threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        fprintf(stdout, "\tMax threads dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0],
                deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        fprintf(stdout, "\tMax grid dimensions: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
                deviceProp.maxGridSize[2]);
        fprintf(stdout, "\tClock rate: %d\n", deviceProp.clockRate);
        fprintf(stdout, "\tTotal constant memory: %ld\n", deviceProp.totalConstMem);
        fprintf(stdout, "\tTexture alignment: %ld\n", deviceProp.textureAlignment);
        fprintf(stdout, "\tConcurrent copy and execution: %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
        fprintf(stdout, "\tNumber of multiprocessors: %d\n", deviceProp.multiProcessorCount);
        fprintf(stdout, "\tKernel execution timeout: %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
        fprintf(stdout, "\tIntegrated: %s\n", deviceProp.integrated ? "Yes" : "No");

        if (i == 0 || bestDeviceProp.totalGlobalMem < deviceProp.totalGlobalMem) {
            bestDevice = i;
            bestDeviceProp = deviceProp;
        }

    }

    fprintf(stdout, "\n################################################\n");
    fprintf(stdout, "Best device: %s (%d)\n", bestDeviceProp.name, bestDevice);

    hipError_t cudaStatus = hipSetDevice(bestDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    size_t *host_k = (size_t *) malloc(sizeof(size_t));
    size_t *dev_k = nullptr;
    *host_k = 3;

    Dataset dataset(argv[1]);

    size_t *host_totalPoints = (size_t *) malloc(sizeof(size_t));
    size_t *dev_totalPoints = nullptr;

    size_t *host_totalLabels = (size_t *) malloc(sizeof(size_t));
    size_t *dev_totalLabels = nullptr;

    *host_totalPoints = dataset.getNPoints();
    *host_totalLabels = dataset.getNLabels();

    if (*host_totalPoints > MAX_GRID_DIM_X) {
        fprintf(stderr, "The number of points is too big for the grid size\n");
        return EXIT_FAILURE;
    }

    DistanceType *host_distanceType = (DistanceType *) malloc(sizeof(DistanceType));
    DistanceType *dev_distanceType = nullptr;

    *host_distanceType = EUCLIDEAN; // This must be corrected

    Point *host_points = dataset.getPoints();
    Point *dev_points = nullptr;

    // N labels to predict
    Label *host_labels = (Label *) malloc(sizeof(Label) * *host_totalLabels);
    Label *dev_labels = nullptr;

    fprintf(stdout, "Total points: %ld.\nResulting points array for training:\n", *host_totalPoints);
    for (size_t i = 0; i < *host_totalPoints; ++i) {
        fprintf(stdout, "%zu: %f %f %f. Label: %zu\n", host_points[i].getId(), host_points[i].getX(),
                host_points[i].getY(),
                host_points[i].getZ(), host_points[i].getLabel());
    }

    fprintf(stdout, "Total labels to predict: \n");
    for (size_t i = 0; i < *host_totalLabels; ++i) {
        host_labels[i].frequency = 0;
        host_labels[i].label = i;
        fprintf(stdout, "%zu: %d.\n", i, host_labels[i].frequency);
    }

    dim3 gpuBlocks(TOTAL_BLOCKS, 1, 1);
    dim3 gpuThreads(*host_totalPoints, 1, 1);

    fprintf(stdout, "################################################\n");
    fprintf(stdout, "GPU blocks: %d, GPU threads: %d\n", gpuBlocks.x, gpuThreads.x);

    // Allocating the points in the GPU
    cudaStatus = hipMalloc((void **) &dev_points, *host_totalPoints * sizeof(Point));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Allocating the labels in the GPU
    cudaStatus = hipMalloc((void **) &dev_labels, *host_totalLabels * sizeof(Label));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Allocating the distance type in the GPU
    cudaStatus = hipMalloc((void **) &dev_distanceType, sizeof(DistanceType));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Allocating the number of points in the GPU
    cudaStatus = hipMalloc((void **) &dev_totalPoints, sizeof(size_t));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Allocating the number of labels in the GPU
    cudaStatus = hipMalloc((void **) &dev_totalLabels, sizeof(size_t));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Allocating the K in the GPU
    cudaStatus = hipMalloc((void **) &dev_k, sizeof(size_t));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the points to the GPU
    cudaStatus = hipMemcpy(dev_points, host_points, *host_totalPoints * sizeof(Point), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the number of points to the GPU
    cudaStatus = hipMemcpy(dev_totalPoints, host_totalPoints, sizeof(size_t), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the number of labels to the GPU
    cudaStatus = hipMemcpy(dev_totalLabels, host_totalLabels, sizeof(size_t), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the distance type to the GPU
    cudaStatus = hipMemcpy(dev_distanceType, host_distanceType, sizeof(DistanceType), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the labels to the GPU
    cudaStatus = hipMemcpy(dev_labels, host_labels, *host_totalLabels * sizeof(Label), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the K to the GPU
    cudaStatus = hipMemcpy(dev_k, host_k, sizeof(size_t), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Creating the distance array into the GPU memory
    double *dev_distances = nullptr;

    cudaStatus = hipMalloc((void **) &dev_distances, *host_totalPoints * sizeof(double));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Creating the query point
    Point *host_queryPoint = (Point *) malloc(sizeof(Point));
    Point *dev_queryPoint = nullptr;

    host_queryPoint->x = 2.5;
    host_queryPoint->y = 7.0;
    host_queryPoint->z = 0.0;

    // Allocating the query point in the GPU
    cudaStatus = hipMalloc((void **) &dev_queryPoint, sizeof(Point));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    // Copying the query point to the GPU
    cudaStatus = hipMemcpy(dev_queryPoint, host_queryPoint, sizeof(Point), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        return EXIT_FAILURE;
    }

    printf("Starting the kernel...\n");

    // Creating a lock
    Lock lock;

    // Creating the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    Label *predictedLabel = (Label *) malloc(sizeof(Label));

    // Starting the kernel
    hipEventRecord(start);
    knn::predict<<<*host_totalPoints, 1>>>(dev_points, dev_totalLabels, dev_totalPoints, dev_k, dev_distanceType, dev_distances,
            dev_queryPoint, dev_labels, lock);

    /*knn::cdp_simple_quicksort<<<*host_totalPoints, 1>>>(dev_distances, dev_points, 0,
            *host_totalPoints - 1, 1); REQUIRED */

    thrust::host_vector<NodeThrust> host_nodesVector(*host_totalPoints);

    // Memcpy the points array to the host
    hipMemcpy(host_points, dev_points, *host_totalPoints * sizeof(Point), hipMemcpyDeviceToHost);

    for (size_t index = 0; index < *host_totalPoints; index++) {
        host_nodesVector[index].x = host_points[index].x;
        host_nodesVector[index].y = host_points[index].y;
        host_nodesVector[index].z = host_points[index].z;
        host_nodesVector[index].label = host_points[index].label;
        host_nodesVector[index].distance = host_points[index].distance;
    }

    thrust::device_vector<NodeThrust> dev_nodesVector = host_nodesVector;

    thrust::sort(dev_nodesVector.begin(), dev_nodesVector.end());

    host_nodesVector = dev_nodesVector;

    for (size_t index = 0; index < *host_k; ++index) {
        for (size_t label = 0; label < *host_totalLabels; ++label) {
            if (host_nodesVector[index].label == label) {
                host_labels[label].frequency = host_labels[label].frequency + 1;
                continue;
            }
        }
    }

    predictedLabel = thrust::max_element(host_labels, host_labels + *host_totalLabels);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, stop);
    printf("Kernel finished. Elapsed time: %f ms\n", miliseconds);
    fprintf(stdout, "################################################\n");

    // Printing the points
    /*
    fprintf(stdout, "Points:\n");
    for (size_t i = 0; i < *host_totalPoints; ++i) {
        fprintf(stdout, "%zu: (%f, %f, %f) --> Label: %zu. Distance to queryPoint: %f\n", i, host_nodesVector[i].x,
                host_nodesVector[i].y, host_nodesVector[i].z, host_nodesVector[i].label, host_nodesVector[i].distance);
    }
     */

    // Printing the labels
    printf("Predicted label: %zu\n", predictedLabel->label);

    return EXIT_SUCCESS;
}